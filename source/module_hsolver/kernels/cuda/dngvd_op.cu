#include "hip/hip_runtime.h"
#include "module_hsolver/kernels/dngvd_op.h"
#include "hip/hip_runtime_api.h"

#include <base/macros/macros.h>

#include <hipsolver.h>

namespace hsolver
{

static hipsolverHandle_t cusolver_H = nullptr;

void createGpuSolverHandle()
{
    if (cusolver_H == nullptr)
    {
        cusolverErrcheck(hipsolverDnCreate(&cusolver_H));
    }
}

void destroyGpuSolverHandle()
{
    if (cusolver_H != nullptr)
    {
        cusolverErrcheck(hipsolverDnDestroy(cusolver_H));
        cusolver_H = nullptr;
    }
}

static inline
void xhegvd_wrapper(
    const hipblasFillMode_t& uplo,
    const int& n,
    double* A, const int& lda,
    double* B, const int& ldb,
    double* W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int* devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    double* work = nullptr;
    cudaErrcheck(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnDsygvd_bufferSize(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
        A, lda, B, ldb, W, &lwork));
    // allocate memery
    cudaErrcheck(hipMalloc((void**)&work, sizeof(double) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnDsygvd(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
        A, lda, B, ldb, W, work, lwork, devInfo));

    cudaErrcheck(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    // free the buffer
    cudaErrcheck(hipFree(work));
    cudaErrcheck(hipFree(devInfo));
}

static inline
void xhegvd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<float> * A, const int& lda,
        std::complex<float> * B, const int& ldb,
        float * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    float2 * work = nullptr;
    cudaErrcheck(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnChegvd_bufferSize(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const float2 *>(A), lda,
                                                 reinterpret_cast<const float2 *>(B), ldb, W, &lwork));
    // allocate memery
    cudaErrcheck(hipMalloc((void**)&work, sizeof(float2) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnChegvd(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<float2 *>(A), lda, reinterpret_cast<float2 *>(B), ldb, W, work, lwork, devInfo));

    cudaErrcheck(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    // free the buffer
    cudaErrcheck(hipFree(work));
    cudaErrcheck(hipFree(devInfo));
}

static inline
void xhegvd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<double> * A, const int& lda,
        std::complex<double> * B, const int& ldb,
        double * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    double2 * work = nullptr;
    cudaErrcheck(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnZhegvd_bufferSize(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const double2 *>(A), lda,
                                                 reinterpret_cast<const double2 *>(B), ldb, W, &lwork));
    // allocate memery
    cudaErrcheck(hipMalloc((void**)&work, sizeof(double2) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZhegvd(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<double2 *>(A), lda, reinterpret_cast<double2 *>(B), ldb, W, work, lwork, devInfo));

    cudaErrcheck(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    // free the buffer
    cudaErrcheck(hipFree(work));
    cudaErrcheck(hipFree(devInfo));
}

static inline
void xheevd_wrapper(
    const hipblasFillMode_t& uplo,
    const int& n,
    double* A, const int& lda,
    double* W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int* devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    double* work = nullptr;
    cudaErrcheck(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnDsyevd_bufferSize(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
        A, lda, W, &lwork));
    // allocate memery
    cudaErrcheck(hipMalloc((void**)&work, sizeof(double) * lwork));
    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnDsyevd(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n, A, lda, W, work, lwork, devInfo));

    cudaErrcheck(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    cudaErrcheck(hipFree(work));
    cudaErrcheck(hipFree(devInfo));
}

static inline
void xheevd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<float> * A, const int& lda,
        float * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    float2 * work = nullptr;
    cudaErrcheck(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnCheevd_bufferSize(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const float2 *>(A), lda, W, &lwork));
    // allocate memery
    cudaErrcheck(hipMalloc((void**)&work, sizeof(float2) * lwork));
    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnCheevd(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n, reinterpret_cast<float2 *>(A), lda, W, work, lwork, devInfo));

    cudaErrcheck(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    cudaErrcheck(hipFree(work));
    cudaErrcheck(hipFree(devInfo));
}

static inline
void xheevd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<double> * A, const int& lda,
        double * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    double2 * work = nullptr;
    cudaErrcheck(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnZheevd_bufferSize(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const double2 *>(A), lda, W, &lwork));
    // allocate memery
    cudaErrcheck(hipMalloc((void**)&work, sizeof(double2) * lwork));
    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZheevd(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<double2 *>(A), lda, W, work, lwork, devInfo));

    cudaErrcheck(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    cudaErrcheck(hipFree(work));
    cudaErrcheck(hipFree(devInfo));
}

template <typename T>
struct dngvd_op<T, base_device::DEVICE_GPU>
{
    using Real = typename GetTypeReal<T>::type;
    void operator()(const base_device::DEVICE_GPU* d,
                    const int nstart,
                    const int ldh,
                    const T* A, // hcc
                    const T* B, // scc
                    Real* W,    // eigenvalue
                    T* V)
    {
        assert(nstart == ldh);
        // A to V
        cudaErrcheck(hipMemcpy(V, A, sizeof(T) * ldh * nstart, hipMemcpyDeviceToDevice));
        xhegvd_wrapper(HIPBLAS_FILL_MODE_UPPER, nstart, V, ldh,
            (T*)B, ldh, W);
    }
};

template <typename T>
struct dnevx_op<T, base_device::DEVICE_GPU>
{
    using Real = typename GetTypeReal<T>::type;
    void operator()(const base_device::DEVICE_GPU* d,
                    const int nstart,
                    const int ldh,
                    const T* A, // hcc
                    const int m,
                    Real* W, // eigenvalue
                    T* V)
    {
        assert(nstart <= ldh);
        // A to V
        cudaErrcheck(hipMemcpy(V, A, sizeof(T) * nstart * ldh, hipMemcpyDeviceToDevice));
        xheevd_wrapper(HIPBLAS_FILL_MODE_LOWER, nstart, V, ldh, W);
    }
};

template <typename T>
struct dngvx_op<T, base_device::DEVICE_GPU>
{
    using Real = typename GetTypeReal<T>::type;
    void operator()(const base_device::DEVICE_GPU* d,
                    const int nbase,
                    const int ldh,
                    T* hcc,
                    T* scc,
                    const int m,
                    Real* eigenvalue,
                    T* vcc)
    {

    }
};

template struct dngvd_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct dnevx_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct dngvx_op<std::complex<float>, base_device::DEVICE_GPU>;

template struct dngvd_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct dnevx_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct dngvx_op<std::complex<double>, base_device::DEVICE_GPU>;

#ifdef __LCAO
template struct dngvd_op<double, base_device::DEVICE_GPU>;
template struct dnevx_op<double, base_device::DEVICE_GPU>;
template struct dngvx_op<double, base_device::DEVICE_GPU>;
#endif

} // namespace hsolver