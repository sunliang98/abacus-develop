#include "hip/hip_runtime.h"
#include "module_base/kernels/math_kernel_op.h"
#include "module_hsolver/kernels/bpcg_kernel_op.h"

#include <base/macros/macros.h>
#include <thrust/complex.h>
#define WARP_SIZE 32
#define THREAD_PER_BLOCK 256
namespace hsolver
{
template <typename Real>
__global__ void line_minimize_with_block(
        thrust::complex<Real>* grad,
        thrust::complex<Real>* hgrad,
        thrust::complex<Real>* psi,
        thrust::complex<Real>* hpsi,
        const int n_basis,
        const int n_basis_max)
{
    int band_idx = blockIdx.x; // band_idx
    int tid = threadIdx.x; // basis_idx
    int item = 0;
    Real epsilo_0 = 0.0, epsilo_1 = 0.0, epsilo_2 = 0.0;
    Real theta = 0.0, cos_theta = 0.0, sin_theta = 0.0;
    __shared__ Real data[THREAD_PER_BLOCK * 3];

    data[tid] = 0;

    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        data[tid] += (grad[item] * thrust::conj(grad[item])).real();
    }
    __syncthreads();
    // just do some parallel reduction in shared memory
    for (int ii = THREAD_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
        }
        __syncthreads();
    }

    Real norm = 1.0 / sqrt(data[0]);
    __syncthreads();

    data[tid] = 0;
    data[THREAD_PER_BLOCK + tid] = 0;
    data[2 * THREAD_PER_BLOCK + tid] = 0;
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        grad[item] *= norm;
        hgrad[item] *= norm;
        data[tid] += (hpsi[item] * thrust::conj(psi[item])).real();
        data[THREAD_PER_BLOCK + tid] += (grad[item] * thrust::conj(hpsi[item])).real();
        data[2 * THREAD_PER_BLOCK + tid] += (grad[item] * thrust::conj(hgrad[item])).real();
    }
    __syncthreads();

    // just do some parallel reduction in shared memory
    for (int ii = THREAD_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
            data[THREAD_PER_BLOCK + tid] += data[THREAD_PER_BLOCK + tid + ii];
            data[2 * THREAD_PER_BLOCK + tid] += data[2 * THREAD_PER_BLOCK + tid + ii];
        }
        __syncthreads();
    }
    epsilo_0 = data[0];
    epsilo_1 = data[THREAD_PER_BLOCK];
    epsilo_2 = data[2 * THREAD_PER_BLOCK];

    theta = 0.5 * abs(atan(2 * epsilo_1/(epsilo_0 - epsilo_2)));
    cos_theta = cos(theta);
    sin_theta = sin(theta);
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        psi [item] = psi [item] * cos_theta + grad [item] * sin_theta;
        hpsi[item] = hpsi[item] * cos_theta + hgrad[item] * sin_theta;
    }
}

template <typename Real>
__global__ void calc_grad_with_block(
        const Real* prec,
        Real* err,
        Real* beta,
        thrust::complex<Real>* psi,
        thrust::complex<Real>* hpsi,
        thrust::complex<Real>* grad,
        thrust::complex<Real>* grad_old,
        const int n_basis,
        const int n_basis_max)
{
    int band_idx = blockIdx.x; // band_idx
    int tid = threadIdx.x; // basis_idx
    int item = 0;
    Real err_st = 0.0;
    Real beta_st = 0.0;
    Real epsilo = 0.0;
    Real grad_2 = 0.0;
    thrust::complex<Real> grad_1 = {0, 0};
    __shared__ Real data[THREAD_PER_BLOCK * 2];

    // Init shared memory
    data[tid] = 0;

    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        data[tid] += (psi[item] * thrust::conj(psi[item])).real();
    }
    __syncthreads();
    // just do some parallel reduction in shared memory
    for (int ii = THREAD_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
        }
        __syncthreads();
    }

    Real norm = 1.0 / sqrt(data[0]);
    __syncthreads();

    data[tid] = 0;
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        psi[item] *= norm;
        hpsi[item] *= norm;
        data[tid] += (hpsi[item] * thrust::conj(psi[item])).real();
    }
    __syncthreads();

    // just do some parallel reduction in shared memory
    for (int ii = THREAD_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
        }
        __syncthreads();
    }
    epsilo = data[0];
    __syncthreads();

    data[tid] = 0;
    data[THREAD_PER_BLOCK + tid] = 0;
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        grad_1 = hpsi[item] - epsilo * psi[item];
        grad_2 = thrust::norm(grad_1);
        data[tid] += grad_2;
        data[THREAD_PER_BLOCK + tid] += grad_2 / prec[basis_idx];
    }
    __syncthreads();

    // just do some parallel reduction in shared memory
    for (int ii = THREAD_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
            data[THREAD_PER_BLOCK + tid] += data[THREAD_PER_BLOCK + tid + ii];
        }
        __syncthreads();
    }
    err_st = data[0];
    beta_st = data[THREAD_PER_BLOCK];
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        grad_1 = hpsi[item] - epsilo * psi[item];
        grad[item] = -grad_1 / prec[basis_idx] + beta_st / beta[band_idx] * grad_old[item];
    }

    __syncthreads();
    if (tid == 0) {
        beta[band_idx] = beta_st;
        err[band_idx] = sqrt(err_st);
    }
}

template <typename Real>
__global__ void apply_eigenvalues_kernel(
        const thrust::complex<Real>* vectors,
        thrust::complex<Real>* result,
        const Real* eigenvalues,
        const int nbase,
        const int nbase_x,
        const int notconv)
{
    int m = blockIdx.x;
    int idx = threadIdx.x + blockIdx.y * blockDim.x;
    
    if (m < notconv && idx < nbase) {
        result[m * nbase_x + idx] = eigenvalues[m] * vectors[m * nbase_x + idx];
    }
}

template <typename Real>
__global__ void precondition_kernel(
        thrust::complex<Real>* psi_iter,
        const Real* precondition,
        const Real* eigenvalues,
        const int dim,
        const int nbase,
        const int notconv)
{
    int m = blockIdx.x;
    int i = threadIdx.x + blockIdx.y * blockDim.x;
    
    if (m < notconv && i < dim) {
        Real x = abs(precondition[i] - eigenvalues[m]);
        Real pre = 0.5 * (1.0 + x + sqrt(1 + (x - 1.0) * (x - 1.0)));
        psi_iter[(nbase + m) * dim + i] = psi_iter[(nbase + m) * dim + i] / pre;
    }
}

template <typename Real>
__global__ void normalize_kernel(
        thrust::complex<Real>* psi_iter,
        Real* psi_norm,
        const int dim,
        const int nbase,
        const int notconv)
{
    int m = blockIdx.x;
    int tid = threadIdx.x;
    __shared__ Real sum[THREAD_PER_BLOCK];
    
    sum[tid] = 0.0;
    
    // Calculate the sum for normalization
    for (int i = tid; i < dim; i += THREAD_PER_BLOCK) {
        auto val = psi_iter[(nbase + m) * dim + i];
        sum[tid] += (val * thrust::conj(val)).real();
    }
    
    __syncthreads();
    
    // Parallel reduction in shared memory
    for (int s = THREAD_PER_BLOCK/2; s > 0; s >>= 1) {
        if (tid < s) {
            sum[tid] += sum[tid + s];
        }
        __syncthreads();
    }
    
    Real norm = sqrt(sum[0]);
    
    // Normalize the vector
    for (int i = tid; i < dim; i += THREAD_PER_BLOCK) {
        psi_iter[(nbase + m) * dim + i] /= norm;
    }
    
    // Store the norm if needed
    if (tid == 0 && psi_norm != nullptr) {
        psi_norm[m] = norm;
    }
}

template <typename T>
void line_minimize_with_block_op<T, base_device::DEVICE_GPU>::operator()(T* grad_out,
                                                                         T* hgrad_out,
                                                                         T* psi_out,
                                                                         T* hpsi_out,
                                                                         const int& n_basis,
                                                                         const int& n_basis_max,
                                                                         const int& n_band)
{
    auto A = reinterpret_cast<thrust::complex<Real>*>(grad_out);
    auto B = reinterpret_cast<thrust::complex<Real>*>(hgrad_out);
    auto C = reinterpret_cast<thrust::complex<Real>*>(psi_out);
    auto D = reinterpret_cast<thrust::complex<Real>*>(hpsi_out);

    line_minimize_with_block<Real><<<n_band, THREAD_PER_BLOCK>>>(
            A, B, C, D,
            n_basis, n_basis_max);

    hipCheckOnDebug();
}

template <typename T>
void calc_grad_with_block_op<T, base_device::DEVICE_GPU>::operator()(const Real* prec_in,
                                                                     Real* err_out,
                                                                     Real* beta_out,
                                                                     T* psi_out,
                                                                     T* hpsi_out,
                                                                     T* grad_out,
                                                                     T* grad_old_out,
                                                                     const int& n_basis,
                                                                     const int& n_basis_max,
                                                                     const int& n_band)
{
    auto A = reinterpret_cast<thrust::complex<Real>*>(psi_out);
    auto B = reinterpret_cast<thrust::complex<Real>*>(hpsi_out);
    auto C = reinterpret_cast<thrust::complex<Real>*>(grad_out);
    auto D = reinterpret_cast<thrust::complex<Real>*>(grad_old_out);

    calc_grad_with_block<Real><<<n_band, THREAD_PER_BLOCK>>>(
            prec_in, err_out, beta_out,
            A, B, C, D,
            n_basis, n_basis_max);

    hipCheckOnDebug();
}

template <typename T>
void apply_eigenvalues_op<T, base_device::DEVICE_GPU>::operator()(const int& nbase,
                                                                const int& nbase_x,
                                                                const int& notconv,
                                                                T* result,
                                                                const T* vectors,
                                                                const Real* eigenvalues)
{
    const int threads_per_block = 256;
    const int blocks_per_grid_y = (nbase + threads_per_block - 1) / threads_per_block;
    
    dim3 grid(notconv, blocks_per_grid_y);
    
    auto vec_complex = reinterpret_cast<const thrust::complex<Real>*>(vectors);
    auto res_complex = reinterpret_cast<thrust::complex<Real>*>(result);
    
    apply_eigenvalues_kernel<Real><<<grid, threads_per_block>>>(
        vec_complex, res_complex, eigenvalues, nbase, nbase_x, notconv);
    
    hipCheckOnDebug();
}

template <typename T>
void precondition_op<T, base_device::DEVICE_GPU>::operator()(const int& dim,
                                                           T* psi_iter,
                                                           const int& nbase,
                                                           const int& notconv,
                                                           const Real* precondition,
                                                           const Real* eigenvalues)
{
    const int threads_per_block = 256;
    const int blocks_per_grid_y = (dim + threads_per_block - 1) / threads_per_block;
    
    dim3 grid(notconv, blocks_per_grid_y);
    
    auto psi_complex = reinterpret_cast<thrust::complex<Real>*>(psi_iter);
    
    precondition_kernel<Real><<<grid, threads_per_block>>>(
        psi_complex, precondition, eigenvalues, dim, nbase, notconv);
    
    hipCheckOnDebug();
}

template <typename T>
void normalize_op<T, base_device::DEVICE_GPU>::operator()(const int& dim,
                                                        T* psi_iter,
                                                        const int& nbase,
                                                        const int& notconv,
                                                        Real* psi_norm)
{
    auto psi_complex = reinterpret_cast<thrust::complex<Real>*>(psi_iter);
    
    normalize_kernel<Real><<<notconv, THREAD_PER_BLOCK>>>(
        psi_complex, psi_norm, dim, nbase, notconv);
    
    hipCheckOnDebug();
}

template struct calc_grad_with_block_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct line_minimize_with_block_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct calc_grad_with_block_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct line_minimize_with_block_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct apply_eigenvalues_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct apply_eigenvalues_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct apply_eigenvalues_op<double, base_device::DEVICE_GPU>;
template struct precondition_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct precondition_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct precondition_op<double, base_device::DEVICE_GPU>;
template struct normalize_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct normalize_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct normalize_op<double, base_device::DEVICE_GPU>;
}