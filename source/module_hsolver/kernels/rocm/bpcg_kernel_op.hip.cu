#include "hip/hip_runtime.h"
#include "module_base/kernels/math_kernel_op.h"
#include "module_hsolver/kernels/bpcg_kernel_op.h"

#include <base/macros/macros.h>
#include <thrust/complex.h>
#define WARP_SIZE 32
#define THREAD_PER_BLOCK 256
namespace hsolver
{
template <typename Real>
__global__ void line_minimize_with_block(
        thrust::complex<Real>* grad,
        thrust::complex<Real>* hgrad,
        thrust::complex<Real>* psi,
        thrust::complex<Real>* hpsi,
        const int n_basis,
        const int n_basis_max)
{
    int band_idx = blockIdx.x; // band_idx
    int tid = threadIdx.x; // basis_idx
    int item = 0;
    Real epsilo_0 = 0.0, epsilo_1 = 0.0, epsilo_2 = 0.0;
    Real theta = 0.0, cos_theta = 0.0, sin_theta = 0.0;
    __shared__ Real data[THREAD_PER_BLOCK * 3];

    data[tid] = 0;

    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        data[tid] += (grad[item] * thrust::conj(grad[item])).real();
    }
    __syncthreads();
    // just do some parallel reduction in shared memory
    for (int ii = THREAD_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
        }
        __syncthreads();
    }

    Real norm = 1.0 / sqrt(data[0]);
    __syncthreads();

    data[tid] = 0;
    data[THREAD_PER_BLOCK + tid] = 0;
    data[2 * THREAD_PER_BLOCK + tid] = 0;
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        grad[item] *= norm;
        hgrad[item] *= norm;
        data[tid] += (hpsi[item] * thrust::conj(psi[item])).real();
        data[THREAD_PER_BLOCK + tid] += (grad[item] * thrust::conj(hpsi[item])).real();
        data[2 * THREAD_PER_BLOCK + tid] += (grad[item] * thrust::conj(hgrad[item])).real();
    }
    __syncthreads();

    // just do some parallel reduction in shared memory
    for (int ii = THREAD_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
            data[THREAD_PER_BLOCK + tid] += data[THREAD_PER_BLOCK + tid + ii];
            data[2 * THREAD_PER_BLOCK + tid] += data[2 * THREAD_PER_BLOCK + tid + ii];
        }
        __syncthreads();
    }
    epsilo_0 = data[0];
    epsilo_1 = data[THREAD_PER_BLOCK];
    epsilo_2 = data[2 * THREAD_PER_BLOCK];

    theta = 0.5 * abs(atan(2 * epsilo_1/(epsilo_0 - epsilo_2)));
    cos_theta = cos(theta);
    sin_theta = sin(theta);
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        psi [item] = psi [item] * cos_theta + grad [item] * sin_theta;
        hpsi[item] = hpsi[item] * cos_theta + hgrad[item] * sin_theta;
    }
}

template <typename Real>
__global__ void calc_grad_with_block(
        const Real* prec,
        Real* err,
        Real* beta,
        thrust::complex<Real>* psi,
        thrust::complex<Real>* hpsi,
        thrust::complex<Real>* grad,
        thrust::complex<Real>* grad_old,
        const int n_basis,
        const int n_basis_max)
{
    int band_idx = blockIdx.x; // band_idx
    int tid = threadIdx.x; // basis_idx
    int item = 0;
    Real err_st = 0.0;
    Real beta_st = 0.0;
    Real epsilo = 0.0;
    Real grad_2 = 0.0;
    thrust::complex<Real> grad_1 = {0, 0};
    __shared__ Real data[THREAD_PER_BLOCK * 2];

    // Init shared memory
    data[tid] = 0;

    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        data[tid] += (psi[item] * thrust::conj(psi[item])).real();
    }
    __syncthreads();
    // just do some parallel reduction in shared memory
    for (int ii = THREAD_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
        }
        __syncthreads();
    }

    Real norm = 1.0 / sqrt(data[0]);
    __syncthreads();

    data[tid] = 0;
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        psi[item] *= norm;
        hpsi[item] *= norm;
        data[tid] += (hpsi[item] * thrust::conj(psi[item])).real();
    }
    __syncthreads();

    // just do some parallel reduction in shared memory
    for (int ii = THREAD_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
        }
        __syncthreads();
    }
    epsilo = data[0];
    __syncthreads();

    data[tid] = 0;
    data[THREAD_PER_BLOCK + tid] = 0;
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        grad_1 = hpsi[item] - epsilo * psi[item];
        grad_2 = thrust::norm(grad_1);
        data[tid] += grad_2;
        data[THREAD_PER_BLOCK + tid] += grad_2 / prec[basis_idx];
    }
    __syncthreads();

    // just do some parallel reduction in shared memory
    for (int ii = THREAD_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
            data[THREAD_PER_BLOCK + tid] += data[THREAD_PER_BLOCK + tid + ii];
        }
        __syncthreads();
    }
    err_st = data[0];
    beta_st = data[THREAD_PER_BLOCK];
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += THREAD_PER_BLOCK) {
        item = band_idx * n_basis_max + basis_idx;
        grad_1 = hpsi[item] - epsilo * psi[item];
        grad[item] = -grad_1 / prec[basis_idx] + beta_st / beta[band_idx] * grad_old[item];
    }

    __syncthreads();
    if (tid == 0) {
        beta[band_idx] = beta_st;
        err[band_idx] = sqrt(err_st);
    }
}

template <typename T>
void line_minimize_with_block_op<T, base_device::DEVICE_GPU>::operator()(T* grad_out,
                                                                         T* hgrad_out,
                                                                         T* psi_out,
                                                                         T* hpsi_out,
                                                                         const int& n_basis,
                                                                         const int& n_basis_max,
                                                                         const int& n_band)
{
    auto A = reinterpret_cast<thrust::complex<Real>*>(grad_out);
    auto B = reinterpret_cast<thrust::complex<Real>*>(hgrad_out);
    auto C = reinterpret_cast<thrust::complex<Real>*>(psi_out);
    auto D = reinterpret_cast<thrust::complex<Real>*>(hpsi_out);

    line_minimize_with_block<Real><<<n_band, THREAD_PER_BLOCK>>>(
            A, B, C, D,
            n_basis, n_basis_max);

    hipCheckOnDebug();
}

template <typename T>
void calc_grad_with_block_op<T, base_device::DEVICE_GPU>::operator()(const Real* prec_in,
                                                                     Real* err_out,
                                                                     Real* beta_out,
                                                                     T* psi_out,
                                                                     T* hpsi_out,
                                                                     T* grad_out,
                                                                     T* grad_old_out,
                                                                     const int& n_basis,
                                                                     const int& n_basis_max,
                                                                     const int& n_band)
{
    auto A = reinterpret_cast<thrust::complex<Real>*>(psi_out);
    auto B = reinterpret_cast<thrust::complex<Real>*>(hpsi_out);
    auto C = reinterpret_cast<thrust::complex<Real>*>(grad_out);
    auto D = reinterpret_cast<thrust::complex<Real>*>(grad_old_out);

    calc_grad_with_block<Real><<<n_band, THREAD_PER_BLOCK>>>(
            prec_in, err_out, beta_out,
            A, B, C, D,
            n_basis, n_basis_max);

    hipCheckOnDebug();
}

template struct calc_grad_with_block_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct line_minimize_with_block_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct calc_grad_with_block_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct line_minimize_with_block_op<std::complex<double>, base_device::DEVICE_GPU>;
}