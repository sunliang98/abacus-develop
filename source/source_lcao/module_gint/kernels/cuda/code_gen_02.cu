#include "vbatch_matrix_mul.cuh"

template void gemm_time_measure<double, 4, 16, 16, 48, 12, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 16, 64, 4, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 16, 64, 8, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 32, 32, 4, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 32, 32, 8, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 32, 32, 12, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 32, 48, 4, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 32, 48, 8, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 48, 32, 4, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 48, 32, 8, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 24, 24, 48, 4, 4, 24, 4, 24>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 24, 24, 48, 8, 4, 24, 4, 24>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 24, 24, 48, 12, 4, 24, 4, 24>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 24, 48, 48, 4, 4, 24, 4, 24>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 24, 48, 48, 8, 4, 24, 4, 24>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 32, 32, 64, 4, 4, 32, 4, 32>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 32, 32, 64, 8, 4, 32, 4, 32>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 32, 32, 64, 12, 4, 32, 4, 32>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 32, 32, 64, 16, 4, 32, 4, 32>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 6, 16, 48, 32, 6, 6, 16, 6, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 6, 16, 48, 32, 12, 6, 16, 6, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 6, 16, 48, 48, 6, 6, 16, 6, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 4, 16, 12, 8, 8, 4, 8, 4>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

