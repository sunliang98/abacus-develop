#include "vbatch_matrix_mul.cuh"

template void gemm_time_measure<double, 8, 16, 32, 32, 24, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 32, 48, 8, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 32, 48, 16, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 32, 64, 8, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 32, 64, 16, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 48, 32, 8, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 48, 32, 16, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 48, 48, 8, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 48, 48, 16, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 48, 64, 8, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 64, 32, 8, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 64, 32, 16, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 16, 64, 48, 8, 8, 16, 8, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 20, 40, 40, 8, 8, 20, 8, 20>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 20, 40, 40, 16, 8, 20, 8, 20>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 20, 40, 60, 8, 8, 20, 8, 20>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 24, 24, 48, 8, 8, 24, 8, 24>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 24, 24, 48, 16, 8, 24, 8, 24>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 24, 24, 48, 24, 8, 24, 8, 24>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 24, 48, 48, 8, 8, 24, 8, 24>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 24, 48, 48, 16, 8, 24, 8, 24>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 28, 56, 56, 8, 8, 28, 8, 28>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 8, 28, 56, 56, 16, 8, 28, 8, 28>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

