#include <omp.h>

#include "kernels/cuda/cuda_tools.cuh"
#include "kernels/cuda/vbatch_matrix_mul.cuh"
#include "module_base/ylm.h"
#include "module_hamilt_lcao/module_gint/gint_tools.h"
#include "module_hamilt_lcao/module_gint/gint_vl.h"
#include "module_hamilt_lcao/module_gint/kernels/cuda/gint_vl.cuh"

namespace GintKernel
{

/**
 * Computes the gamma component of the VL (Vlocal) integral on the GPU.
 *
 * @param hRGint Pointer to the HContainer<double> object to store the computed
 * integrals.
 * @param lgd Dimension information for the computation results.
 * @param max_size The maximum number of neighboring atoms for a grid point.
 * @param vfactor Related to volume. The scaling factor for the Vlocal
 * integrals.
 * @param vlocal Pointer to the Vlocal array.
 * @param ylmcoef_now Pointer to the Ylm coefficients array.
 * @param nczp The number of grid layers in the C direction.
 * @param nbxx The total number of grid points.
 * @param dr The grid spacing.
 * @param rcut Pointer to the cutoff radius array.
 * @param gridt The Grid_Technique object containing grid information.
 * @param ucell The UnitCell object containing unit cell information.
 *
 * @note The grid integration on the GPU is mainly divided into the following
 * steps:
 * 1. Use the CPU to divide the grid integration into subtasks.
 * 2. Copy the subtask information to the GPU.
 * 3. Calculate the matrix elements on the GPU.
 * 4. Perform matrix multiplication on the GPU.
 * 5. Copy the results back to the host.
 */
void gint_gamma_vl_gpu(hamilt::HContainer<double>* hRGint,
                       const int lgd,
                       const int max_size,
                       const double vfactor,
                       const double* vlocal,
                       const double* ylmcoef_now,
                       const int nczp,
                       const int nbxx,
                       const double dr,
                       const double* rcut,
                       const Grid_Technique& gridt,
                       const UnitCell& ucell)
{
    const int nbz = gridt.nbzp;
    checkCuda(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
    {
        int iter_num = 0;
        for (int iat1 = 0; iat1 < ucell.nat; iat1++)
        {
            for (int iat2 = 0; iat2 < ucell.nat; iat2++)
            {
                int stream_num = iter_num % gridt.nstreams;
                int it1 = ucell.iat2it[iat1];
                int lo1 = gridt.trace_lo[ucell.itiaiw2iwt(it1,
                                                          ucell.iat2ia[iat1],
                                                          0)];

                int it2 = ucell.iat2it[iat2];
                int lo2 = gridt.trace_lo[ucell.itiaiw2iwt(it2,
                                                          ucell.iat2ia[iat2],
                                                          0)];

                if (lo1 <= lo2)
                {
                    hamilt::AtomPair<double>* tmp_ap
                        = hRGint->find_pair(iat1, iat2);
                    if (tmp_ap == nullptr)
                    {
                        continue;
                    }
                    int atom_pair_nw
                        = ucell.atoms[it1].nw * ucell.atoms[it2].nw;
                    if (gridt.grid_vlocal_g[iat1 * ucell.nat + iat2] == nullptr)
                    {
                        checkCuda(hipMallocAsync(
                            (void**)&gridt
                                .grid_vlocal_g[iat1 * ucell.nat + iat2],
                            atom_pair_nw * sizeof(double),
                            gridt.streams[stream_num]));
                    }
                    checkCuda(hipMemsetAsync(
                        gridt.grid_vlocal_g[iat1 * ucell.nat + iat2],
                        0,
                        atom_pair_nw * sizeof(double),
                        gridt.streams[stream_num]));
                    iter_num++;
                }
            }
        }
    }
    for (int i = 0; i < gridt.nstreams; i++)
    {
        checkCuda(hipStreamSynchronize(gridt.streams[i]));
    }

#pragma omp parallel for num_threads(gridt.nstreams) collapse(2)
    for (int i = 0; i < gridt.nbx; i++)
    {
        for (int j = 0; j < gridt.nby; j++)
        {
            int stream_num = omp_get_thread_num();
            checkCuda(hipStreamSynchronize(gridt.streams[stream_num]));
            double* input_double
                = &gridt.psi_dbl_gbl[gridt.psi_size_max * stream_num * 5];
            int* input_int
                = &gridt.psi_int_gbl[gridt.psi_size_max * stream_num * 2];
            int* num_psir = &gridt.num_psir_gbl[nbz * stream_num];
            int* atom_pair_A_m
                = &gridt.l_info_global[gridt.atom_pair_nbz * stream_num];
            int* atom_pair_B_n
                = &gridt.r_info_global[gridt.atom_pair_nbz * stream_num];
            int* atom_pair_k
                = &gridt.k_info_global[gridt.atom_pair_nbz * stream_num];
            int* atom_pair_lda
                = &gridt.lda_info_global[gridt.atom_pair_nbz * stream_num];
            int* atom_pair_ldb
                = &gridt.ldb_info_global[gridt.atom_pair_nbz * stream_num];
            int* atom_pair_ldc
                = &gridt.ldc_info_global[gridt.atom_pair_nbz * stream_num];

            double* input_double_g
                = &gridt.psi_dbl_gbl_g[gridt.psi_size_max * stream_num * 5];
            int* input_int_g
                = &gridt.psi_int_gbl_g[gridt.psi_size_max * stream_num * 2];
            int* num_psir_g = &gridt.num_psir_gbl_g[nbz * stream_num];
            double* psir_ylm_left_g
                = &gridt.left_global_g[gridt.psir_size * stream_num];
            double* psir_r_g
                = &gridt.right_global_g[gridt.psir_size * stream_num];

            int* atom_pair_A_m_g
                = &gridt.l_info_global_g[gridt.atom_pair_nbz * stream_num];
            int* atom_pair_B_n_g
                = &gridt.r_info_global_g[gridt.atom_pair_nbz * stream_num];
            int* atom_pair_k_g
                = &gridt.k_info_global_g[gridt.atom_pair_nbz * stream_num];

            int* atom_pair_lda_g
                = &gridt.lda_info_gbl_g[gridt.atom_pair_nbz * stream_num];
            int* atom_pair_ldb_g
                = &gridt.ldb_info_gbl_g[gridt.atom_pair_nbz * stream_num];
            int* atom_pair_ldc_g
                = &gridt.ldc_info_gbl_g[gridt.atom_pair_nbz * stream_num];

            double** matrix_A
                = &gridt.ap_left_gbl[gridt.atom_pair_nbz * stream_num];
            double** matrix_B
                = &gridt.ap_right_gbl[gridt.atom_pair_nbz * stream_num];
            double** matrix_C
                = &gridt.ap_output_gbl[gridt.atom_pair_nbz * stream_num];

            double** matrix_A_g
                = &gridt.ap_left_gbl_g[gridt.atom_pair_nbz * stream_num];
            double** matrix_B_g
                = &gridt.ap_right_gbl_g[gridt.atom_pair_nbz * stream_num];
            double** matrix_C_g
                = &gridt.ap_output_gbl_g[gridt.atom_pair_nbz * stream_num];
            int atom_pair_num = 0;
            int max_m = 0;
            int max_n = 0;

            gtask_vlocal(gridt,
                         rcut,
                         ucell,
                         i,
                         j,
                         max_size,
                         nczp,
                         vfactor,
                         vlocal,
                         psir_ylm_left_g,
                         psir_r_g,
                         input_double,
                         input_int,
                         num_psir,
                         atom_pair_A_m,
                         atom_pair_B_n,
                         atom_pair_lda,
                         atom_pair_ldb,
                         atom_pair_ldc,
                         matrix_A,
                         matrix_B,
                         matrix_C,
                         atom_pair_num,
                         max_m,
                         max_n);

            for (int z = 0; z < gridt.atom_pair_nbz; z++)
            {
                atom_pair_k[z] = gridt.bxyz;
            }

            checkCuda(hipMemcpyAsync(input_double_g,
                                      input_double,
                                      gridt.psi_size_max * 5 * sizeof(double),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));
            checkCuda(hipMemcpyAsync(input_int_g,
                                      input_int,
                                      gridt.psi_size_max * 2 * sizeof(int),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));
            checkCuda(hipMemcpyAsync(num_psir_g,
                                      num_psir,
                                      nbz * sizeof(int),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));

            checkCuda(hipMemcpyAsync(atom_pair_A_m_g,
                                      atom_pair_A_m,
                                      gridt.atom_pair_nbz * sizeof(int),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));
            checkCuda(hipMemcpyAsync(atom_pair_B_n_g,
                                      atom_pair_B_n,
                                      gridt.atom_pair_nbz * sizeof(int),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));
            checkCuda(hipMemcpyAsync(atom_pair_k_g,
                                      atom_pair_k,
                                      gridt.atom_pair_nbz * sizeof(int),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));
            checkCuda(hipMemcpyAsync(atom_pair_lda_g,
                                      atom_pair_lda,
                                      gridt.atom_pair_nbz * sizeof(int),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));
            checkCuda(hipMemcpyAsync(atom_pair_ldb_g,
                                      atom_pair_ldb,
                                      gridt.atom_pair_nbz * sizeof(int),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));
            checkCuda(hipMemcpyAsync(atom_pair_ldc_g,
                                      atom_pair_ldc,
                                      gridt.atom_pair_nbz * sizeof(int),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));

            checkCuda(hipMemcpyAsync(matrix_A_g,
                                      matrix_A,
                                      gridt.atom_pair_nbz * sizeof(double*),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));
            checkCuda(hipMemcpyAsync(matrix_B_g,
                                      matrix_B,
                                      gridt.atom_pair_nbz * sizeof(double*),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));

            checkCuda(hipMemcpyAsync(matrix_C_g,
                                      matrix_C,
                                      gridt.atom_pair_nbz * sizeof(double*),
                                      hipMemcpyHostToDevice,
                                      gridt.streams[stream_num]));

            checkCuda(hipMemsetAsync(psir_ylm_left_g,
                                      0,
                                      gridt.psir_size * sizeof(double),
                                      gridt.streams[stream_num]));
            checkCuda(hipMemsetAsync(psir_r_g,
                                      0,
                                      gridt.psir_size * sizeof(double),
                                      gridt.streams[stream_num]));

            dim3 grid_psi(nbz, 8);
            dim3 block_psi(64);

            get_psi_and_vldr3<<<grid_psi,
                                block_psi,
                                0,
                                gridt.streams[stream_num]>>>(
                gridt.ylmcoef_g,
                dr,
                gridt.bxyz,
                ucell.nwmax,
                input_double_g,
                input_int_g,
                num_psir_g,
                gridt.psi_size_max_z,
                gridt.atom_nwl_g,
                gridt.atom_new_g,
                gridt.atom_ylm_g,
                gridt.atom_nw_g,
                gridt.nr_max,
                gridt.psi_u_g,
                psir_ylm_left_g,
                psir_r_g);
            checkCudaLastError();
            gridt.fastest_matrix_mul(max_m,
                                     max_n,
                                     atom_pair_A_m_g,
                                     atom_pair_B_n_g,
                                     atom_pair_k_g,
                                     matrix_A_g,
                                     atom_pair_lda_g,
                                     matrix_B_g,
                                     atom_pair_ldb_g,
                                     matrix_C_g,
                                     atom_pair_ldc_g,
                                     atom_pair_num,
                                     gridt.streams[stream_num],
                                     nullptr);
            // checkCuda(hipStreamSynchronize(gridt.streams[stream_num]));
        }
    }
    for (int i = 0; i < gridt.nstreams; i++)
    {
        checkCuda(hipStreamSynchronize(gridt.streams[i]));
    }
    {
        int iter_num = 0;
        for (int iat1 = 0; iat1 < ucell.nat; iat1++)
        {
            for (int iat2 = 0; iat2 < ucell.nat; iat2++)
            {
                int stream_num = iter_num % gridt.nstreams;
                int it1 = ucell.iat2it[iat1];
                int lo1 = gridt.trace_lo[ucell.itiaiw2iwt(it1,
                                                          ucell.iat2ia[iat1],
                                                          0)];

                int it2 = ucell.iat2it[iat2];
                int lo2 = gridt.trace_lo[ucell.itiaiw2iwt(it2,
                                                          ucell.iat2ia[iat2],
                                                          0)];
                if (lo1 <= lo2)
                {
                    int atom_pair_nw
                        = ucell.atoms[it1].nw * ucell.atoms[it2].nw;
                    hamilt::AtomPair<double>* tmp_ap
                        = hRGint->find_pair(iat1, iat2);
                    if (tmp_ap == nullptr)
                    {
                        continue;
                    }
                    checkCuda(hipMemcpyAsync(
                        tmp_ap->get_pointer(0),
                        gridt.grid_vlocal_g[iat1 * ucell.nat + iat2],
                        atom_pair_nw * sizeof(double),
                        hipMemcpyDeviceToHost,
                        gridt.streams[stream_num]));
                    iter_num++;
                }
            }
        }
    }
    for (int i = 0; i < gridt.nstreams; i++)
    {
        checkCuda(hipStreamSynchronize(gridt.streams[i]));
    }
}

} // namespace GintKernel