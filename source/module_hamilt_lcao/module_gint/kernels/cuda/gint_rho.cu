#include "hip/hip_runtime.h"
#include "interp.cuh"
#include "module_hamilt_lcao/module_gint/kernels/cuda/gint_rho.cuh"
#include "sph.cuh"

namespace GintKernel
{

__global__ void get_psi(const double* const ylmcoef,
                        double delta_r_g,
                        int bxyz_g,
                        double nwmax_g,
                        const double* const input_double,
                        const int* const input_int,
                        const int* const num_psir,
                        int psi_size_max,
                        const int* const ucell_atom_nwl,
                        const bool* const atom_iw2_new,
                        const int* const atom_iw2_ylm,
                        const int* const atom_nw,
                        int nr_max,
                        const double* const psi_u,
                        double* psir_ylm)
{
    int size = num_psir[blockIdx.x];
    int start_index = psi_size_max * blockIdx.x;
    int end_index = start_index + size;
    start_index += threadIdx.x + blockDim.x * blockIdx.y;
    for (int index = start_index; index < end_index;
         index += blockDim.x * gridDim.y)
    {
        double dr[3];
        int index_double = index * 5;
        dr[0] = input_double[index_double];
        dr[1] = input_double[index_double + 1];
        dr[2] = input_double[index_double + 2];
        double distance = input_double[index_double + 3];
        double ylma[49];
        int index_int = index * 2;
        int it = input_int[index_int];
        int dist_tmp = input_int[index_int + 1];
        int nwl = ucell_atom_nwl[it];

        spherical_harmonics(dr, distance, nwl, ylma, ylmcoef);

        interpolate(distance,
                    delta_r_g,
                    it,
                    nwmax_g,
                    nr_max,
                    atom_nw,
                    atom_iw2_new,
                    psi_u,
                    ylma,
                    atom_iw2_ylm,
                    psir_ylm,
                    dist_tmp,
                    1);
    }
}

__global__ void psir_dot(const int* n,
                         double** vec_l_g,
                         int incl,
                         double** vec_r_g,
                         int incr,
                         double** results_g,
                         int batchcount)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = id; i < batchcount; i += stride)
    {
        double* sum = results_g[i];
        double* x = vec_l_g[i];
        double* y = vec_r_g[i];

        for (int j = 0; j < n[i]; j++)
        {
            sum[0] += x[j * incl] * y[j * incr];
        }
    }
}

} // namespace GintKernel