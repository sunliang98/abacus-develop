#include "vbatch_matrix_mul.cuh"

template void gemm_time_measure<double, 4, 8, 16, 32, 8, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 16, 40, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 16, 48, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 16, 56, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 24, 16, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 24, 16, 8, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 24, 24, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 24, 24, 8, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 24, 32, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 24, 40, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 32, 16, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 32, 16, 8, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 32, 24, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 40, 16, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 40, 24, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 48, 16, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 8, 56, 16, 4, 4, 8, 4, 8>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 16, 32, 4, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 16, 32, 8, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 16, 32, 12, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 16, 32, 16, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 16, 48, 4, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

template void gemm_time_measure<double, 4, 16, 16, 48, 8, 4, 16, 4, 16>(int,int,int*,int*,int*,double**,int*,double**,int*,double**,int*,int,hipStream_t,float&,matrix_multiple_func_type&,double*,double*,double*);

