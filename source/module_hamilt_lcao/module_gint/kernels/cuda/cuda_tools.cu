#include <iostream>

#include "module_hamilt_lcao/module_gint/kernels/cuda/cuda_tools.cuh"
hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}
hipError_t checkCudaLastError()
{
    hipError_t result = hipGetLastError();
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

void dump_cuda_array_to_file(double* cuda_array,
                             int width,
                             int hight,
                             const std::string& filename)
{
    double* h_data = new double[width * hight];
    hipMemcpy(h_data,
               cuda_array,
               width * hight * sizeof(double),
               hipMemcpyDeviceToHost);

    std::ofstream outFile(filename);
    if (!outFile.is_open())
    {
        std::cerr << "Failed to open file for writing." << std::endl;
    }
    for (int j = 0; j < hight; ++j)
    {
        for (int i = 0; i < width; ++i)
        {
            outFile << "hight" << j << "   width:" << i << "   "
                    << h_data[j * width + i] << std::endl;
        }
    }
    outFile.close();
    delete[] h_data;
}

template <typename T>
Cuda_Mem_Wrapper<T>::Cuda_Mem_Wrapper(int one_stream_size_in,
                                      int one_stream_size_aligned_in,
                                      int stream_number_in,
                                      bool malloc_host_in)
{
    this->stream_number = stream_number_in;
    this->one_stream_size = one_stream_size_in;
    this->one_stream_size_aligned = one_stream_size_aligned_in;
    this->total_size_aligned
        = this->one_stream_size_aligned * this->stream_number;

    checkCuda(hipMalloc((void**)&this->device_pointer,
                         this->total_size_aligned * sizeof(T)));
    checkCuda(hipMemset(this->device_pointer,
                         0,
                         this->total_size_aligned * sizeof(T)));
    this->host_pointer = nullptr;

    if (malloc_host_in)
    {
        checkCuda(hipHostMalloc((void**)&this->host_pointer,
                                 this->total_size_aligned * sizeof(T)));
        memset(this->host_pointer, 0, this->total_size_aligned * sizeof(T));
    }
}
template <typename T>
Cuda_Mem_Wrapper<T>::Cuda_Mem_Wrapper(int one_stream_size_in,
                                      int stream_number_in,
                                      bool malloc_host_in)
    : Cuda_Mem_Wrapper(one_stream_size_in,
                       one_stream_size_in,
                       stream_number_in,
                       malloc_host_in)
{
}
template <typename T>
void Cuda_Mem_Wrapper<T>::free_all()
{
    checkCuda(hipFree(this->device_pointer));
    if (this->host_pointer != nullptr)
    {
        checkCuda(hipHostFree(this->host_pointer));
    }
}

template <typename T>
Cuda_Mem_Wrapper<T>::~Cuda_Mem_Wrapper()
{
    this->free_all();
}

template <typename T>
void Cuda_Mem_Wrapper<T>::copy_host_to_device_sync(int stream_id)
{
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy host to device"
                  << std::endl;
        exit(1);
    }
    checkCuda(hipMemcpy(
        this->device_pointer + stream_id * this->one_stream_size_aligned,
        this->host_pointer + stream_id * this->one_stream_size_aligned,
        this->one_stream_size * sizeof(T),
        hipMemcpyHostToDevice));
}

template <typename T>
void Cuda_Mem_Wrapper<T>::copy_host_to_device_async(hipStream_t stream,
                                                    int stream_id)
{
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy host to device"
                  << std::endl;
        exit(1);
    }
    checkCuda(hipMemcpyAsync(
        this->device_pointer + stream_id * this->one_stream_size_aligned,
        this->host_pointer + stream_id * this->one_stream_size_aligned,
        this->one_stream_size * sizeof(T),
        hipMemcpyHostToDevice,
        stream));
}

template <typename T>
void Cuda_Mem_Wrapper<T>::copy_device_to_host_sync(int stream_id)
{
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy device to host"
                  << std::endl;
        exit(1);
    }
    checkCuda(hipMemcpy(
        this->host_pointer + stream_id * this->one_stream_size_aligned,
        this->device_pointer + stream_id * this->one_stream_size_aligned,
        this->one_stream_size * sizeof(T),
        hipMemcpyDeviceToHost));
}

template <typename T>
void Cuda_Mem_Wrapper<T>::copy_device_to_host_async(hipStream_t stream,
                                                    int stream_id)
{
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy device to host"
                  << std::endl;
        exit(1);
    }
    checkCuda(hipMemcpyAsync(
        this->host_pointer + stream_id * this->one_stream_size_aligned,
        this->device_pointer + stream_id * this->one_stream_size_aligned,
        this->one_stream_size * sizeof(T),
        hipMemcpyDeviceToHost,
        stream));
}
template <typename T>
void Cuda_Mem_Wrapper<T>::memset_device_sync(int stream_id, int value)
{
    checkCuda(hipMemset(this->device_pointer
                             + stream_id * this->one_stream_size_aligned,
                         value,
                         this->one_stream_size * sizeof(T)));
}

template <typename T>
void Cuda_Mem_Wrapper<T>::memset_device_async(hipStream_t stream,
                                              int stream_id,
                                              int value)
{
    checkCuda(hipMemsetAsync(this->device_pointer
                                  + stream_id * this->one_stream_size_aligned,
                              value,
                              this->one_stream_size * sizeof(T),
                              stream));
}

template <typename T>
void Cuda_Mem_Wrapper<T>::memset_host(int stream_id, int value)
{
    memset(this->host_pointer + stream_id * this->one_stream_size_aligned,
           value,
           this->one_stream_size * sizeof(T));
}

template <typename T>
T* Cuda_Mem_Wrapper<T>::get_device_pointer(int stream_id)
{
    return this->device_pointer + stream_id * this->one_stream_size_aligned;
}

template <typename T>
T* Cuda_Mem_Wrapper<T>::get_host_pointer(int stream_id)
{
    return this->host_pointer + stream_id * this->one_stream_size_aligned;
}
template class Cuda_Mem_Wrapper<double>;
template class Cuda_Mem_Wrapper<double*>;
template class Cuda_Mem_Wrapper<int>;